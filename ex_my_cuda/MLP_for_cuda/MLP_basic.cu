#include "hip/hip_runtime.h"
#include "MLP_basic.h"

#define IDX2C(i,j,Id)       (((j)*(Id))+(i)) // j -> row, i -> column



using namespace std;


float GausianRandom(float average, float stdev) 
{
    double v1, v2, s, temp;

    do {
        v1 =  2 * ((float) rand() / RAND_MAX) - 1;      // -1.0 ~ 1.0 까지의 값
        v2 =  2 * ((float) rand() / RAND_MAX) - 1;      // -1.0 ~ 1.0 까지의 값
        s = v1 * v1 + v2 * v2;
    } while (s >= 1 || s == 0);

    s = sqrt( (-2 * log(s)) / s );

    temp = v1 * s;
    temp =( stdev*temp) + average;


    return temp;
}

MLP_basic:: MLP_basic()
{
    total_layers = 0;
    mini_batch = 0;
    alpha = 0;
    ramda = 0;
    target = NULL;
    d_target = NULL;
    d_temp = NULL;
    d_temp1 = NULL;
    d_one_vector = NULL;
    input = NULL; 
    for(int i = 0 ; i < MAXIMUM_LAYERS ; i++)
    {
        neural[i] = 0;
        W[i] = NULL;
        b[i] = NULL;

        d_W[i] = NULL;
        d_b[i] = NULL;
        d_a[i] = NULL;
        d_z[i] = NULL;
        d_delta[i] = NULL;
        d_delta_W[i] = NULL;
        d_delta_b[i] = NULL;

    }

    hipblasCreate(&handle);
}

MLP_basic :: ~MLP_basic()
{
   if(target != NULL) free(target);
   if(d_target != NULL) hipFree(d_target);
   if(d_temp != NULL) hipFree(d_temp);
   if(d_temp1 != NULL) hipFree(d_temp1);
   if(d_one_vector != NULL) hipFree(d_one_vector);
   if(input != NULL) free(input);

   for(int i = 0 ; i < MAXIMUM_LAYERS ; i++)
   {
       if(W[i] != NULL) free(W[i]);
       if(b[i] != NULL) free(b[i]);
         
       if(d_W[i] != NULL) hipFree(d_W[i]);
       if(d_b[i] != NULL) hipFree(d_b[i]);
       if(d_a[i] != NULL) hipFree(d_a[i]);
       if(d_z[i] != NULL) hipFree(d_z[i]);
       if(d_delta[i] != NULL) hipFree(d_delta[i]);
       if(d_delta_W[i] != NULL) hipFree(d_delta_W[i]);
       if(d_delta_b[i] != NULL) hipFree(d_delta_b[i]);
   }
   hipblasDestroy(handle);
    
    
}

void MLP_basic :: init(int *neurals,int layers,int batch_size,float alpha, float ramda)
{
    this->total_layers = layers;
    this->mini_batch = batch_size;
    this->alpha = alpha;
    this->ramda = ramda;
    
    for(int i = 0 ; i < this->total_layers ; i++)
    {
        this->neural[i] = neurals[i];
    }

    hipMalloc(&d_target,sizeof(float)*neural[total_layers-1]*mini_batch);
    hipMalloc(&d_a[0],sizeof(float)*neural[0]*mini_batch);

    target = (float*)calloc(neural[total_layers-1]*mini_batch,sizeof(float));
    input = (float*)calloc(neural[0]*mini_batch,sizeof(float));
    

    int maximum = 0;
    for(int i = 0 ; i < total_layers-1 ; i++)
    {
        W[i] = (float*)calloc(neural[i]*neural[i+1],sizeof(float));
        b[i] = (float*)calloc(neural[i+1],sizeof(float));

        hipMalloc(&d_W[i],sizeof(float)*neural[i]*neural[i+1]);
        hipMalloc(&d_b[i],sizeof(float)*neural[i+1]);
        hipMalloc(&d_a[i+1],sizeof(float)*neural[i+1]*mini_batch);
        hipMalloc(&d_z[i+1],sizeof(float)*neural[i+1]*mini_batch);
        hipMalloc(&d_delta[i+1],sizeof(float)*neural[i+1]*mini_batch);
        hipMalloc(&d_delta_W[i],sizeof(float)*neural[i+1]*neural[i]);
        hipMalloc(&d_delta_b[i],sizeof(float)*neural[i+1]);
        if(neural[i] > maximum) maximum = neural[i]; 
    } 
    hipMalloc(&d_temp,sizeof(float)*maximum*maximum); //temp alloc
    hipMalloc(&d_temp1,sizeof(float)*maximum*maximum);
    
    float *one_vector;
    one_vector = (float*)calloc(mini_batch,sizeof(float));
    for(int i = 0 ; i < mini_batch ; i++) one_vector[i] = 1.0;
    
    hipMalloc(&d_one_vector,sizeof(float)*mini_batch);
    hipblasSetMatrix(1,mini_batch,sizeof(float),one_vector,1,d_one_vector,1);  
    
    free(one_vector);

}

void MLP_basic :: test_example()
{
   input[0] = 0;
    input[1] = 0;
    input[2] = 0;
    input[3] = 0;
    input[4] = 0;
    input[5] = 1;
    input[6] = 0;
    input[7] = 1;
    input[8] = 0;
    input[9] = 0;
    input[10] = 1;
    input[11] = 1;
    input[12] = 1;
    input[13] = 0;
    input[14] = 0;
    input[15] = 1;
    input[16] = 0;
    input[17] = 1;
    input[18] = 1;
    input[19] = 1;
    input[20] = 0;
    input[21] = 1;
    input[22] = 1;
    input[23] = 1;

    target[0] = 1;
    target[1] = 1;
    target[2] = 1;
    target[3] = 0;
    target[4] = 0;
    target[5] = 0;
    target[6] = 0;
    target[7] = 1;
    target[8] = 0;
    target[9] = 1;
    target[10] = 0;
    target[11] = 0;
    target[12] = 1;
    target[13] = 0;
    target[14] = 1;
    target[15] = 1;


        
    W[0][0] = -0.2052;
    W[0][1] = 0.3735;
    W[0][2] = -0.2398;
    W[0][3] = -0.3509;
    W[0][4] = -0.2674;
    W[0][5] = -0.1811;
    W[0][6] = -0.1767;
    W[0][7] = 0.0564;
    W[0][8] = -0.0623;
    W[0][9] = 0.1854;
    W[0][10] = -0.2694;
    W[0][11] = 0.2531;
   
    W[1][0] = -0.6215;
    W[1][1] = -0.4016;
    W[1][2] = 0.3716;
    W[1][3] = 0.6417;
    W[1][4] = -0.0870;
    W[1][5] = 0.1909;
    W[1][6] = 0.5633;
    W[1][7] = 0.5609;
    W[1][8] = -0.5898;
    W[1][9] = -0.3324;
    W[1][10] = -0.1320;
    W[1][11] = 0.4434;

    W[2][0] = 0.0209;
    W[2][1] = 0.1427;
    W[2][2] = -0.1019;
    W[2][3] = 0.2493;
    W[2][4] = 0.2937;
    W[2][5] = -0.1552;

    b[0][0] = -0.4773;
    b[0][1] = 0.2770;
    b[0][2] = -0.0768;
    b[0][3] = 0.0528;

    b[1][0] = -0.4325;
    b[1][1] = -0.0925;
    b[1][2] = 0.4019;

    b[2][0] = 0.5683;
    b[2][1] = 0.1868;


}

void MLP_basic :: cpy_host_device()
{

    for(int i = 0 ; i < total_layers -1 ; i++)
    {
        hipblasSetMatrix(neural[i+1],neural[i],sizeof(float),W[i],neural[i+1],d_W[i],neural[i+1]);
        hipblasSetVector(neural[i+1],sizeof(float),b[i],1,d_b[i],1); 
    }

    //test code
    hipblasSetMatrix(neural[0],mini_batch,sizeof(float),input,neural[0],d_a[0],neural[0]);    
    hipblasSetMatrix(neural[total_layers-1],mini_batch,sizeof(float),target,neural[total_layers-1],d_target,neural[total_layers-1]);
}


__global__ void add_bias(float *z,float *b,int column,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  while(tid < n)
  {
      z[tid] += b[tid % column];  
      
      tid+= blockDim.x * gridDim.x;
  }
}
__global__ void sigmoid(float *a,float *z,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  while(tid < n)
  {
      a[tid] = 1/(1+expf(-z[tid]));
      
      tid+= blockDim.x * gridDim.x;
  }
}




void MLP_basic :: activation()
{
    float one = 1.0;
    float zero = 0.0;
    int threadsPerBolck = 1024;
    int blocksPerGride = 0; 
    


    //z2 = W1*a1
    hipblasSgemm(handle,  HIPBLAS_OP_N,HIPBLAS_OP_N,neural[1],mini_batch,neural[0],  &one,  d_W[0],neural[1],  d_a[0],neural[0],  &zero,  d_z[1],neural[1]);
    //z2 = z2 + b1;
    blocksPerGride = (neural[1]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    add_bias<<<blocksPerGride, threadsPerBolck>>>(d_z[1],d_b[0],neural[1],neural[1]*mini_batch);
    //a2 = F(z2)
    blocksPerGride = (neural[1]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    sigmoid<<<blocksPerGride, threadsPerBolck>>>(d_a[1],d_z[1],neural[1]*mini_batch);
    //

    //z3 = W2*a2
    hipblasSgemm(handle,  HIPBLAS_OP_N,HIPBLAS_OP_N,neural[2],mini_batch,neural[1],  &one,  d_W[1],neural[2],  d_a[1],neural[1],  &zero,  d_z[2],neural[2]);
    //z3 = z3 + b2;
    blocksPerGride = (neural[2]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    add_bias<<<blocksPerGride, threadsPerBolck>>>(d_z[2],d_b[1],neural[2],neural[2]*mini_batch);
    //a3 = F(z3)
    blocksPerGride = (neural[2]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    sigmoid<<<blocksPerGride, threadsPerBolck>>>(d_a[2],d_z[2],neural[2]*mini_batch);
    //
    
    //z4 = W3*a3
    hipblasSgemm(handle,  HIPBLAS_OP_N,HIPBLAS_OP_N,neural[3],mini_batch,neural[2],  &one,  d_W[2],neural[3],  d_a[2],neural[2],  &zero,  d_z[3],neural[3]);
    //z4 = z4 + b3;
    blocksPerGride = (neural[3]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    add_bias<<<blocksPerGride, threadsPerBolck>>>(d_z[3],d_b[2],neural[3],neural[3]*mini_batch);
    //a4 = F(z4)
    blocksPerGride = (neural[3]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    sigmoid<<<blocksPerGride, threadsPerBolck>>>(d_a[3],d_z[3],neural[3]*mini_batch);
    
/*
    hipblasGetMatrix(neural[3],mini_batch,sizeof(float),d_a[3],neural[3],target,neural[3]);
    
    for(int y = 0 ; y < neural[3] ; y++)
    {
        for(int x = 0 ; x < mini_batch ;x++)
        {
            cout<<target[IDX2C(y,x,neural[3])]<<" ";
        }
        cout<<endl;
    }
    */
}

__global__ void last_delta_before_transpose(float *temp, float *y,float *T,int batch_size,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  while(tid < n)
  {
      temp[tid] = (y[tid]-T[tid])/(2*batch_size);   
      tid+= blockDim.x * gridDim.x;
  }
}
__global__ void transpose(float *after, float *before,int before_columns,int before_rows)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int x,y;
  
  while(tid < before_columns*before_rows)
  {
      y = tid % before_columns;
      x = tid / before_columns;
      after[IDX2C(x,y,before_rows)] = before[IDX2C(y,x,before_columns)];
      tid+= blockDim.x * gridDim.x;
  }
}

__global__ void sigmoid_inv(float *a,float *z,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  while(tid < n)
  {
      a[tid] = (1/(1+expf(-z[tid])))*(1 - 1/(1+expf(-z[tid])));
      tid+= blockDim.x * gridDim.x;
  }
}

__global__ void basic_multi(float *a,float *b,float *c, int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  while(tid < n)
  {
      c[tid] = a[tid]*b[tid]; 
      tid+= blockDim.x * gridDim.x;
  }
}



void MLP_basic :: delta_rule()
{
    float one = 1.0;
    float zero = 0.0;
    int threadsPerBolck = 1024;
    int blocksPerGride = 0; 

   

    // temp = (y-T)*(2*batch_size)
    blocksPerGride = (neural[3]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    last_delta_before_transpose<<<blocksPerGride, threadsPerBolck>>>(d_temp,d_a[3],d_target,mini_batch,neural[3]*mini_batch);      
    //delta4 = transpose(temp)
    blocksPerGride = (neural[3]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_delta[3],d_temp,neural[3],mini_batch);
    
  
    //delta3 = delta4*W3
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,mini_batch,neural[2],neural[3],  &one,  d_delta[3],mini_batch,  d_W[2],neural[3],  &zero,  d_delta[2],mini_batch);  
    //temp = f_inv(z3)
    blocksPerGride = (neural[2]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    sigmoid_inv<<<blocksPerGride, threadsPerBolck>>>(d_temp,d_z[2],neural[2]*mini_batch);   
    //temp1 = transpose(temp) 
    blocksPerGride = (neural[2]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_temp1,d_temp,neural[2],mini_batch);
    //delta3 = delta3.*temp1
    blocksPerGride = (neural[2]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    basic_multi<<<blocksPerGride, threadsPerBolck>>>(d_delta[2],d_temp1,d_delta[2],neural[2]*mini_batch);
    

    //delta2 = delta3*W2
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,mini_batch,neural[1],neural[2],  &one,  d_delta[2],mini_batch,  d_W[1],neural[2],  &zero,  d_delta[1],mini_batch);
    //temp = f_inv(z2)
    blocksPerGride = (neural[1]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    sigmoid_inv<<<blocksPerGride, threadsPerBolck>>>(d_temp,d_z[1],neural[1]*mini_batch);
    //temp1 = transpose(temp) 
    blocksPerGride = (neural[1]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_temp1,d_temp,neural[1],mini_batch);
    //delta2 = delta2.*temp1
    blocksPerGride = (neural[1]*mini_batch + threadsPerBolck -1)/threadsPerBolck;
    basic_multi<<<blocksPerGride, threadsPerBolck>>>(d_delta[1],d_temp1,d_delta[1],neural[1]*mini_batch);
    
      /* 
    
    float aaa[10000];
    hipblasGetMatrix(mini_batch,neural[1],sizeof(float),d_delta[1],mini_batch,aaa,mini_batch);
    
    for(int y = 0 ; y < mini_batch ; y++)
    {
        for(int x = 0 ; x < neural[1] ;x++)
        {
            cout<<aaa[IDX2C(y,x,mini_batch)]<<" ";
        }
        cout<<endl;
    }
   cout<<endl; 
    */
      
}



__global__ void weight_update(float *w,float *delta_w, float alpha,float ramda,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
 
  while(tid < n)
  {
      w[tid] = w[tid] - alpha*(delta_w[tid] + ramda*w[tid]);    
      tid+= blockDim.x * gridDim.x;
  }
}
__global__ void bias_update(float *b,float *delta_b, float alpha,int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
 
  while(tid < n)
  {
      b[tid] = b[tid] - alpha*delta_b[tid];    
      tid+= blockDim.x * gridDim.x;
  }
}




void MLP_basic :: update()
{
    float one = 1.0;
    float zero = 0.0;
    int threadsPerBolck = 1024;
    int blocksPerGride = 0; 
    

    
    //temp = a3*delta4
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,neural[2],neural[3],mini_batch,  &one,  d_a[2],neural[2],  d_delta[3],mini_batch,  &zero,  d_temp,neural[2]);  
    //delta_W3 = transpose(temp)
    blocksPerGride = (neural[2]*neural[3] + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_delta_W[2],d_temp,neural[2],neural[3]);
    //W3 = W3 - alpha*(delta_W3 + ramda*W3) 
    blocksPerGride = (neural[3]*neural[2] + threadsPerBolck -1)/threadsPerBolck;
    weight_update<<<blocksPerGride, threadsPerBolck>>>(d_W[2],d_delta_W[2],alpha,ramda,neural[3]*neural[2]);   
   
    //temp = a2*delta3
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,neural[1],neural[2],mini_batch,  &one,  d_a[1],neural[1],  d_delta[2],mini_batch,  &zero,  d_temp,neural[1]);  
    //delta_W2 = transpose(temp)
    blocksPerGride = (neural[1]*neural[2] + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_delta_W[1],d_temp,neural[1],neural[2]);
    //W2 = W2 - alpha*(delta_W2 + ramda*W2) 
    blocksPerGride = (neural[2]*neural[1] + threadsPerBolck -1)/threadsPerBolck;
    weight_update<<<blocksPerGride, threadsPerBolck>>>(d_W[1],d_delta_W[1],alpha,ramda,neural[2]*neural[1]);
    
    //temp = a1*delta2
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,neural[0],neural[1],mini_batch,  &one,  d_a[0],neural[0],  d_delta[1],mini_batch,  &zero,  d_temp,neural[0]);  
    //delta_W1 = transpose(temp)
    blocksPerGride = (neural[0]*neural[1] + threadsPerBolck -1)/threadsPerBolck;
    transpose<<<blocksPerGride, threadsPerBolck>>>(d_delta_W[0],d_temp,neural[0],neural[1]);
    //W1 = W1 - alpha*(delta_W1 + ramda*W1) 
    blocksPerGride = (neural[1]*neural[0] + threadsPerBolck -1)/threadsPerBolck;
    weight_update<<<blocksPerGride, threadsPerBolck>>>(d_W[0],d_delta_W[0],alpha,ramda,neural[1]*neural[0]);
  
    
    
    //delta_b3 = one_vector*delta4
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,neural[3],mini_batch,  &one,  d_one_vector,1,  d_delta[3],mini_batch,  &zero,  d_delta_b[2],1);  
    //b3 = b3 - alpha*transpose(delta_b3)
    blocksPerGride = (neural[3] + threadsPerBolck -1)/threadsPerBolck;
    bias_update<<<blocksPerGride, threadsPerBolck>>>(d_b[2],d_delta_b[2],alpha,neural[3]);   

    //delta_b2 = one_vector*delta3
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,neural[2],mini_batch,  &one,  d_one_vector,1,  d_delta[2],mini_batch,  &zero,  d_delta_b[1],1);  
    //b2 = b2 - alpha*transpose(delta_b2)
    blocksPerGride = (neural[2] + threadsPerBolck -1)/threadsPerBolck;
    bias_update<<<blocksPerGride, threadsPerBolck>>>(d_b[1],d_delta_b[1],alpha,neural[2]);   

    //delta_b1 = one_vector*delta2
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,neural[1],mini_batch,  &one,  d_one_vector,1,  d_delta[1],mini_batch,  &zero,  d_delta_b[0],1);  
    //b1 = b1 - alpha*transpose(delta_b1)
    blocksPerGride = (neural[1] + threadsPerBolck -1)/threadsPerBolck;
    bias_update<<<blocksPerGride, threadsPerBolck>>>(d_b[0],d_delta_b[0],alpha,neural[1]);   

/*
    float bbb[10000];  
    hipblasStatus_t stat; 
    stat = hipblasGetMatrix(1,neural[1],sizeof(float),d_b[0],1,bbb,1);
  
    cout<<stat<<endl;

    for(int y = 0 ; y < mini_batch ; y++)
    {
        cout<<bbb[y]<<" ";
    }
    cout<<endl; 

*/


/*  
    float aaa[10000];  
    hipblasStatus_t stat; 
    stat = hipblasGetMatrix(neural[1],neural[0],sizeof(float),d_W[0],neural[1],aaa,neural[1]);
  
    cout<<stat<<endl;

    for(int y = 0 ; y < neural[1] ; y++)
    {
        for(int x = 0 ; x < neural[0] ;x++)
        {
            cout<<aaa[IDX2C(y,x,neural[1])]<<" ";
        }
        cout<<endl;
    }
    cout<<endl; 

 */
   


}























