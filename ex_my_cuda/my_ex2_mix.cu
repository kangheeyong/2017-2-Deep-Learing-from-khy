#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>

//보통 행렬은 3*4 행렬이면 
//
// a11 a12 a13 a14
// a21 a22 a23 a24
// a31 a32 a33 a34
//
// a11 a12 a13 a14 a21 a22 a23 a24 a31 a32 a33 a34 
// 위와 같이 저장하지만
//
// cuBLAS에서는
// 
// a11 a21 a31 a12 a22 a32 a13 a23 a33 a14 a24 a34
// 위와 같이 저장된다.
//
// 보통 열(세로,column)의 수  기준으로 저장하지만
// cuBLAS에서는 행(가로, row)의 수 기준으로 저장한다.
//

#define IDX2C(i,j,Id)       (((j)*(Id))+(i)) // j -> row, i -> column

using namespace std;


bool ChoseGpuAvailable(int n)
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);


    cout<<"devicesCount : "<<devicesCount<<endl;
    
    for(int i = 0 ; i < devicesCount ; i++)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties,i);
        cout<<"----- device "<<i<<" -----"<<endl;
        cout<<"device name : "<<deviceProperties.name<<endl;
        cout<<"maxThreadsPerBlock : "<<deviceProperties.maxThreadsPerBlock<<endl;
        cout<<"warpSize : "<<deviceProperties.warpSize<<endl;

    }
    if(n > devicesCount && n < 0) return false;
    else
    {
        hipSetDevice(n);

        return true;
    }
}

__global__ void my_kernel(int m, int k, int n, float *a,float *b, float *c)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int x,y;
    
  while(tid < m*n)
  {
      y = tid % n;
      x = tid / n;
      c[IDX2C(x,y,m)] = 0.0;
      for(int i = 0 ; i < k ;i++)
      {
          c[IDX2C(x,y,m)] += a[IDX2C(y,i,n)]*b[IDX2C(i,x,k)];
      }
      tid+= blockDim.x * gridDim.x;
  }
}



int main(int argc, char** argv)
{

    unsigned int m = 6; //row
    unsigned int n = 4; //column
    unsigned int k = 5;
    float *matrix1, *matrix2, *matrix3;
    clock_t t;
    int host2device_time, device2host_time,GPU_time;

    matrix1 = new float[n*k];
    matrix2 = new float[k*m];
    matrix3 = new float[m*n];
    
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    ChoseGpuAvailable(1);
    
     //데이터 초기화
   
    int ind = 11;

    for(int x = 0 ; x < k ; x++)
    {
        for(int y = 0 ; y < n ; y++)
        {
            matrix1[IDX2C(y,x,n)] = (float)ind++;
            
        }
    }
    ind = 11;
    for(int x = 0 ; x < m ; x++)
    {
        for(int y = 0 ; y < k ; y++)
        {
            matrix2[IDX2C(y,x,k)] = (float)ind++;
            
        }
    }
    ind = 11;
    for(int x = 0 ; x < n ; x++)
    {
        for(int y = 0 ; y < m ; y++)
        {
            matrix3[IDX2C(y,x,m)] = (float)ind++;
            
        }
    }



    cout<<"maxtrix1"<<endl;
    for(int y = 0 ; y < n ; y++)
    {
        for(int x = 0 ; x < k ; x++)
        {
            cout<<matrix1[IDX2C(y,x,n)]<<" "; 
            
        }
        cout<<endl;
    }
     cout<<"maxtrix2"<<endl;
    for(int y = 0 ; y < k ; y++)
    {
        for(int x = 0 ; x < m ; x++)
        {
            cout<<matrix2[IDX2C(y,x,k)]<<" "; 
            
        }
        cout<<endl;
    }
    cout<<"maxtrix3"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix3[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
    
    cout<<endl;


    //cuda 메모리 할당
    float *d_matrix1, *d_matrix2, *d_matrix3;
    hipMalloc(&d_matrix1,n*k*sizeof(float));
    hipMalloc(&d_matrix2,k*m*sizeof(float));
    hipMalloc(&d_matrix3,m*n*sizeof(float));

    // memory -> cuda memory
    t = clock();
    hipblasCreate(&handle);
    hipblasSetMatrix(n,k,sizeof(float),matrix1,n,d_matrix1,n);
    hipblasSetMatrix(k,m,sizeof(float),matrix2,k,d_matrix2,k);
    hipblasSetMatrix(m,n,sizeof(float),matrix3,m,d_matrix3,m);
    host2device_time = clock()-t;


    // 연산 (커널 실행)
    float al=1.0f;
    float bet=0.0f;
    t = clock();
    //stat = cublasSgemv(handle,  CUBLAS_OP_N,m,n,  &al,  d_matrix1,m,  d_vector1,1,  &bet,  d_vector2,1);
    //
    //먼저 주소 값의 바꿔주면서 행렬의 (0,0)의 위치를 바꿔주고
    //3,4번째 파라메터로 행렬의 최종 크기를 정해준다.
    //
    //CUBLAS_OP_N은 아무것도 안한것
    //CUBLAS_OP_T는 transpose한것이다.
    //
    //
    int threadsPerBolck = 1024;
    int blocksPerGride = (m*n + threadsPerBolck -1)/threadsPerBolck;
    my_kernel<<<blocksPerGride, threadsPerBolck>>>(m,k,n,d_matrix1,d_matrix2,d_matrix3);
  

    GPU_time = clock() - t;
    //cuda memory -> memory
    
    t= clock();
     
    hipblasGetMatrix(n,k,sizeof(float),d_matrix1,n,matrix1,n);
    hipblasGetMatrix(k,m,sizeof(float),d_matrix2,k,matrix2,k);
    hipblasGetMatrix(m,n,sizeof(float),d_matrix3,m,matrix3,m);
 
    device2host_time = clock() - t; 
     //결과 확인
    

    cout<<"maxtrix1"<<endl;
    for(int y = 0 ; y < n ; y++)
    {
        for(int x = 0 ; x < k ; x++)
        {
            cout<<matrix1[IDX2C(y,x,n)]<<" "; 
            
        }
        cout<<endl;
    }
     cout<<"maxtrix2"<<endl;
    for(int y = 0 ; y < k ; y++)
    {
        for(int x = 0 ; x < m ; x++)
        {
            cout<<matrix2[IDX2C(y,x,k)]<<" "; 
            
        }
        cout<<endl;
    }
    cout<<"maxtrix3"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix3[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
    

    
   
    cout<<"host to device time : "<<host2device_time<<endl;
    cout<<"GPU time : "<<GPU_time<<endl;
    cout<<"device to host time : "<<device2host_time<<endl;
   
    //cuda 메모리 해제
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
    
    hipblasDestroy(handle);

    delete matrix1;
    delete matrix2;
    delete matrix3;


    return 0;
}

