#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>

//보통 행렬은 3*4 행렬이면 
//
// a11 a12 a13 a14
// a21 a22 a23 a24
// a31 a32 a33 a34
//
// a11 a12 a13 a14 a21 a22 a23 a24 a31 a32 a33 a34 
// 위와 같이 저장하지만
//
// cuBLAS에서는
// 
// a11 a21 a31 a12 a22 a32 a13 a23 a33 a14 a24 a34
// 위와 같이 저장된다.
//
// 보통 열(세로,column)의 수  기준으로 저장하지만
// cuBLAS에서는 행(가로, row)의 수 기준으로 저장한다.
//

#define IDX2C(i,j,Id)       (((j)*(Id))+(i)) // j -> row, i -> column

using namespace std;


bool ChoseGpuAvailable(int n)
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);


    cout<<"devicesCount : "<<devicesCount<<endl;
    
    for(int i = 0 ; i < devicesCount ; i++)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties,i);
        cout<<"----- device "<<i<<" -----"<<endl;
        cout<<"device name : "<<deviceProperties.name<<endl;
        cout<<"maxThreadsPerBlock : "<<deviceProperties.maxThreadsPerBlock<<endl;
        cout<<"warpSize : "<<deviceProperties.warpSize<<endl;

    }
    if(n > devicesCount && n < 0) return false;
    else
    {
        hipSetDevice(n);

        return true;
    }
}



int main(int argc, char** argv)
{

    unsigned int m = 6; //row
    unsigned int n = 4; //column
    unsigned int k = 5;
    float *matrix1, *matrix2, *matrix3;
    clock_t t;
    int host2device_time, device2host_time,GPU_time;

    matrix1 = new float[m*k];
    matrix2 = new float[k*n];
    matrix3 = new float[m*n];
    
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    
    ChoseGpuAvailable(1);
    
     //데이터 초기화
   
    int ind = 11;

    for(int x = 0 ; x < k ; x++)
    {
        for(int y = 0 ; y < m ; y++)
        {
            matrix1[IDX2C(y,x,m)] = (float)ind++;
            
        }
    }
    ind = 11;
    for(int x = 0 ; x < n ; x++)
    {
        for(int y = 0 ; y < k ; y++)
        {
            matrix2[IDX2C(y,x,k)] = (float)ind++;
            
        }
    }
    ind = 11;
    for(int x = 0 ; x < n ; x++)
    {
        for(int y = 0 ; y < m ; y++)
        {
            matrix3[IDX2C(y,x,m)] = (float)ind++;
            
        }
    }



    cout<<"maxtrix1"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < k ; x++)
        {
            cout<<matrix1[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
     cout<<"maxtrix2"<<endl;
    for(int y = 0 ; y < k ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix2[IDX2C(y,x,k)]<<" "; 
            
        }
        cout<<endl;
    }
    cout<<"maxtrix3"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix3[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
    
    cout<<endl;


    //cuda 메모리 할당
    float *d_matrix1, *d_matrix2, *d_matrix3;
    hipMalloc(&d_matrix1,m*k*sizeof(float));
    hipMalloc(&d_matrix2,k*n*sizeof(float));
    hipMalloc(&d_matrix3,m*n*sizeof(float));

    // memory -> cuda memory
    t = clock();
    hipblasCreate(&handle);
    hipblasSetMatrix(m,k,sizeof(float),matrix1,m,d_matrix1,m);
    hipblasSetMatrix(k,n,sizeof(float),matrix2,k,d_matrix2,k);
    hipblasSetMatrix(m,n,sizeof(float),matrix3,m,d_matrix3,m);
    host2device_time = clock()-t;


    // 연산 (커널 실행)
    float al=1.0f;
    float bet=0.0f;
    t = clock();
    //stat = cublasSgemv(handle,  CUBLAS_OP_N,m,n,  &al,  d_matrix1,m,  d_vector1,1,  &bet,  d_vector2,1);
    //
    //먼저 주소 값의 바꿔주면서 행렬의 (0,0)의 위치를 바꿔주고
    //3,4번째 파라메터로 행렬의 최종 크기를 정해준다.
    //
    //CUBLAS_OP_N은 아무것도 안한것
    //CUBLAS_OP_T는 transpose한것이다.
    //
    //
    stat = hipblasSgemm(handle,  HIPBLAS_OP_T,HIPBLAS_OP_N,m-4,n-2,k-3,  &al,  d_matrix1+m,m,  d_matrix2,k,  &bet,  d_matrix3,m);
    

    GPU_time = clock() - t;
    //cuda memory -> memory
    
    t= clock();
     
    hipblasGetMatrix(m,k,sizeof(float),d_matrix1,m,matrix1,m);
    hipblasGetMatrix(k,n,sizeof(float),d_matrix2,k,matrix2,k);
    hipblasGetMatrix(m,n,sizeof(float),d_matrix3,m,matrix3,m);
 
    device2host_time = clock() - t; 
     //결과 확인
    

    cout<<"maxtrix1"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < k ; x++)
        {
            cout<<matrix1[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
     cout<<"maxtrix2"<<endl;
    for(int y = 0 ; y < k ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix2[IDX2C(y,x,k)]<<" "; 
            
        }
        cout<<endl;
    }
    cout<<"maxtrix3"<<endl;
    for(int y = 0 ; y < m ; y++)
    {
        for(int x = 0 ; x < n ; x++)
        {
            cout<<matrix3[IDX2C(y,x,m)]<<" "; 
            
        }
        cout<<endl;
    }
    

    
   
    cout<<"host to device time : "<<host2device_time<<endl;
    cout<<"GPU time : "<<GPU_time<<endl;
    cout<<"device to host time : "<<device2host_time<<endl;
   
    //cuda 메모리 해제
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
    
    hipblasDestroy(handle);

    delete matrix1;
    delete matrix2;
    delete matrix3;


    return 0;
}

